#include "init.h"
#include "kernel.h"

#include "errors.h"

#ifdef CUDA
int* kernel_main_cuda(int device, const char* pixels, int w, int h)
{
    int* ret = (int*)malloc(sizeof(int)*3*MAX_ERRORS);
    memset((char*)ret, '\0', sizeof(int)*3*MAX_ERRORS);

    hipSetDevice(device);

    char* devPixels;
    int* error_buffer;
    hipMalloc((void**)&devPixels, w*h);
    hipMalloc((void**)&error_buffer, sizeof(int)*3*MAX_ERRORS);

    hipMemset(error_buffer, 0, sizeof(int)*3*MAX_ERRORS);

    hipMemcpy(devPixels, pixels, w*h, hipMemcpyHostToDevice);

    /* NOTES:
     * 1. Warps (blocks) come in multiples of 32, so make sure your block size
     *    is a multiple of 32, or else the remaining SMP cores will be wasted.
     * 2. Each thread is limited to MAX_ERRORS/<numthreads> errors, so there's
     *    no memory overflow. If you need more errors than that, your file is
     *    too buggy and it's not my fault.
     * 3. Each thread schedules it's own checks using just it's thread ID and
     *    the dimensions of the image. All threads do all horizontal checks, and
     *    then all move on to vertical checks at the same time.
     * 4. Each thread does the row in the image that corresponds with its thread
     *    ID, then increments rows by the number of threads until it reaches the
     *    end. The same is done for vertical checks.
     * 5. The maximum thread count is 65,535, but I highly doubt it's possible to
     *    write something that can use that many threads without deadlocking.
     *    Plus this usecase has no need for that.
     */

    //dim3 blocks(32);
    //dim3 threads(6, 6);

    device_drc<<<64, 1>>>(devPixels, w, h, error_buffer);

    hipMemcpy(ret, error_buffer, sizeof(int)*3*MAX_ERRORS, hipMemcpyDeviceToHost);
    hipFree(error_buffer);
    hipFree(devPixels);

    return ret;
}
#else
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>

int* kernel_main_cpu(const char* pixels, int w, int h)
{
    int* ret = (int*)malloc(sizeof(int)*3*MAX_ERRORS);
    memset((char*)ret, '\0', sizeof(int)*3*MAX_ERRORS);

    cpu_drc(pixels, w, h, ret);
    return ret;
}
#endif
