#include "init.h"
#include "kernel.h"
#include "errors.h"

#include <stdio.h>

#ifdef CUDA
void handle_malloc(hipError_t err, size_t size, const char *file, int line) {
    if(err != hipSuccess) {
        printf("hipMalloc failed: %s (tried to malloc %d bytes) in %s at line %d\n",
               hipGetErrorString(err), size, file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_MALLOC(err, size) (handle_malloc(err, size, __FILE__, __LINE__))

int* kernel_main_cuda(int device, const char* pixels, int w, int h, int blocks, int threads)
{
    hipSetDevice(device);

    char* devPixels;
    int* error_buffer;

    size_t memFree;
    size_t memTot;
    hipMemGetInfo(&memFree, &memTot);
    if(w*h > memFree) {
        printf("Not enough device memory available: need %d, available %d (total %d)\n",
               w*h, memFree, memTot);
        exit(EXIT_FAILURE);
    }

    HANDLE_MALLOC(hipMalloc((void**)&devPixels, w*h), w*h);
    HANDLE_MALLOC(hipMalloc((void**)&error_buffer, sizeof(int)*3*MAX_ERRORS), w*h);
    hipMemset(error_buffer, 0, sizeof(int)*3*MAX_ERRORS);
    hipMemcpy(devPixels, pixels, w*h, hipMemcpyHostToDevice);

    /* NOTES:
     * 1. Warps (blocks) come in multiples of 32, so make sure your block size
     *    is a multiple of 32, or else the remaining SMP cores will be wasted.
     * 2. Each thread is limited to MAX_ERRORS/<numthreads> errors, so there's
     *    no memory overflow. If you need more errors than that, your file is
     *    too buggy and it's not my fault.
     * 3. Each thread schedules it's own checks using just it's thread ID and
     *    the dimensions of the image. All threads do all horizontal checks, and
     *    then all move on to vertical checks at the same time.
     * 4. Each thread does the row in the image that corresponds with its thread
     *    ID, then increments rows by the number of threads until it reaches the
     *    end. The same is done for vertical checks.
     * 5. The maximum thread count is 65,535, but I highly doubt it's possible to
     *    write something that can use that many threads without deadlocking.
     *    Plus this usecase has no need for that.
     */

    device_drc<<<blocks, threads>>>(devPixels, w, h, error_buffer);
    hipDeviceSynchronize();

    int* ret = (int*)malloc(sizeof(int)*3*MAX_ERRORS);
    hipMemcpy(ret, error_buffer, sizeof(int)*3*MAX_ERRORS, hipMemcpyDeviceToHost);

    hipFree(error_buffer);
    hipFree(devPixels);

    return ret;
}
#else
#include <stdlib.h>
#include <memory.h>

#include <QElapsedTimer>
#include <QString>

int* kernel_main_cpu(const char* pixels, int w, int h)
{
    int* ret = (int*)malloc(sizeof(int)*3*MAX_ERRORS);
    memset((char*)ret, '\0', sizeof(int)*3*MAX_ERRORS);

    cpu_drc(pixels, w, h, ret);

    return ret;
}
#endif
